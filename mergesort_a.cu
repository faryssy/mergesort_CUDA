
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

typedef struct
{
int x;
int y;
} Point;

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

__global__ void mergeSmall_k(int *A, int *B, int *C, int n)
{   
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = 0;
    Point Q = {0,0};
    Point K = {0,0};
    Point P = {0,0};

    if (idx > sizeof(A))
    {
       K = {idx - (int)sizeof(A), (int)sizeof(A)};
       P = {(int)sizeof(A), idx - (int)sizeof(A)};
    }
    else
    {
        K = {0, idx};
        P = {idx, 0};
    }

    while (true) 
    {
        offset = abs(K.y - P.y)/2;
        Q.x = K.x + offset;
        Q.y = K.y - offset;
        
        if (Q.y >= 0 && Q.x <= sizeof(B) && 
            (Q.y == sizeof(A) || Q.x == 0 || A[Q.y] > B[Q.x - 1]))
            {
                if (Q.x == sizeof(B) || Q.y == 0 || A[Q.y - 1] <= B[Q.x])
                {
                    if (Q.y < sizeof(A) && (Q.x == sizeof(B) || A[Q.y] <= B[Q.x]))
                    {
                        C[idx] = A[Q.y];
                    }

                    else
                    {
                        C[idx] = B[Q.x];
                    }
                    break;
                }
                else
                {
                    K.x = Q.x + 1;
                    K.y = Q.y - 1;
                }
            }
        else 
        {
            P.x = Q.x - 1;
            P.y = Q.y + 1;
        }
    }   
    __syncthreads();
}

int main(void)
{
   // GPU timer instructions
    float TimeExec;									
	hipEvent_t start, stop;						
	testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));	

    // size of A and B
    const int n = 8;

    // host arrays
    int *A, *B, *C;

    // device arrays
    int *d_A, *d_B, *d_C;

    // Allocate memory for host arrays
    A = (int*)malloc(n*sizeof(int));
    B = (int*)malloc(n*sizeof(int));
    C = (int*)malloc(2*n*sizeof(int));

    // Allocate memory for device arrays
    hipMalloc(&d_A, n*sizeof(int));
    hipMalloc(&d_B, n*sizeof(int));
    hipMalloc(&d_C, 2*n*sizeof(int));

    // Initialize arrays on host
    for (int i = 0; i < n*2+1; i++)
    {
      if (i%2)
      {
        A[i/2] = i;
      }
      else
      {
        B[(i-1)/2] = i;
      }
    }
    
    testCUDA(hipEventRecord(start, 0));

    // Copy host vectors to device
    hipMemcpy( d_A, A, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_B, B, n*sizeof(int), hipMemcpyHostToDevice);
    // Executing the kernel function
    mergeSmall_k<<<1, 2*n>>>(d_A, d_B, d_C, n);

    // Copy array back to host
    hipMemcpy( C, d_C, 2*n*sizeof(int), hipMemcpyDeviceToHost);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));			
    testCUDA(hipEventElapsedTime(&TimeExec, start, stop));							
    testCUDA(hipEventDestroy(start));				
    testCUDA(hipEventDestroy(stop));

    // print C = mergeSmall_k(A,B)
    printf("Mergesort of A and B \n");
    printf("A : ");
    for(int i = 0; i < n; i++)
        printf("%d ", A[i]);
    printf("\nB : ");
    for(int i = 0; i < n; i++)
        printf("%d ", B[i]);
    
    printf("\n");
    for(int i = 0; i < 2*n; i++)
        printf("%d ", C[i]);
    
    printf("\nGPU time execution for Merge Sort: %f ms\n", TimeExec);

    // Release device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Release host memory
    free(A);
    free(B);
    free(C);
 
    return 0;
}
